#include "hip/hip_runtime.h"
﻿#define GLM_FORCE_CUDA

#include <hip/hip_runtime.h>
#include "kernel.h"
#include "utilityCore.hpp"

#include <cmath>
#include <cstdio>
#include <iostream>
#include <vector>

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>

#include <glm/glm.hpp>

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3* dev_sortedPos;
glm::vec3* dev_sortedVel1;
glm::vec3* dev_sortedVel2;
// thrust for the above buffers
thrust::device_ptr<glm::vec3> dev_thrust_particlePosIndices;
thrust::device_ptr<glm::vec3> dev_thrust_particleVel1Indices;


// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;
// extra param added
float maxRuleDist;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  maxRuleDist = std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  gridCellWidth = 2.0f * maxRuleDist;
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");
  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  // 2.3 buffers
  hipMalloc((void**)&dev_sortedPos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_sortedPos failed!");
  hipMalloc((void**)&dev_sortedVel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_sortedVel1 failed!");
  hipMalloc((void**)&dev_sortedVel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_sortedVel2 failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

__device__ glm::vec3 rule1(int N, int iSelf, const glm::vec3* pos)
{
    glm::vec3 perceived_center;         // 0, 0, 0 atm, but maybe should be smth else?
    int num_neighbors = 0;

    for (int i = 0; i < N; ++i)
    {
        if (i != iSelf && glm::distance(pos[i], pos[iSelf]) < rule1Distance)
        {
            perceived_center += pos[i];
            ++num_neighbors;
        }
    }
    if (num_neighbors == 0) return glm::vec3(0);
    perceived_center /= num_neighbors;
    return (perceived_center - pos[iSelf]) * rule1Scale;
}

__device__ glm::vec3 rule2(int N, int iSelf, const glm::vec3* pos)
{
    glm::vec3 c;
    for (int i = 0; i < N; ++i)
    {
        if (iSelf != i && glm::distance(pos[i], pos[iSelf]) < rule2Distance)
        {
            c -= (pos[i] - pos[iSelf]);
        }
    }

    return c * rule2Scale;
}

__device__ glm::vec3 rule3(int N, int iSelf, const glm::vec3* pos, const glm::vec3* vel)
{
    glm::vec3 perceived_vel;
    int num_neighbors = 0;

    for (int i = 0; i < N; ++i)
    {
        if (i != iSelf && glm::distance(pos[i], pos[iSelf]) < rule3Distance)
        {
            perceived_vel += vel[i];
            ++num_neighbors;
        }
    }
    if (num_neighbors == 0) return glm::vec3(0);
    perceived_vel /= num_neighbors;
    return perceived_vel * rule3Scale;
}

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
    glm::vec3 vel_change(0);
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    vel_change += rule1(N, iSelf, pos);
  // Rule 2: boids try to stay a distance d away from each other
    vel_change += rule2(N, iSelf, pos);
  // Rule 3: boids try to match the speed of surrounding boids
    vel_change += rule3(N, iSelf, pos, vel);
  return vel_change;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3* pos,
    glm::vec3* vel1, glm::vec3* vel2) {
    // Compute a new velocity based on pos and vel1
    // Clamp the speed
    // Record the new velocity into vel2. Question: why NOT vel1?
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    glm::vec3 new_vel = vel1[index] + computeVelocityChange(N, index, pos, vel1);
    new_vel = new_vel.length() > maxSpeed ? glm::normalize(new_vel) * maxSpeed : new_vel;
    vel2[index] = new_vel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N)
    {
        indices[index] = index;
        // we care about relative position from the start of the grid, then bucket based on cell width
        glm::vec3 relativePos = (pos[index] - gridMin) * inverseCellWidth;
        gridIndices[index] = gridIndex3Dto1D(relativePos.x, relativePos.y, relativePos.z, gridResolution);
    }
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
   int index = (blockIdx.x * blockDim.x) + threadIdx.x;
   if (index >= N) return;
   int thisGrid = particleGridIndices[index];

   if (index > 0 && index < N - 1 && thisGrid != particleGridIndices[index - 1])
   {
       gridCellStartIndices[thisGrid] = index;
       gridCellEndIndices[thisGrid - 1] = index - 1;
   }
   else if (index == 0)
   {
       gridCellStartIndices[thisGrid] = index;
   }
   else if (index == N - 1)
   {
       gridCellEndIndices[thisGrid] = index;
   }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2, float maxRuleDist) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
    int tIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tIdx >= N) return;
    int currBoid = particleArrayIndices[tIdx];
    glm::vec3 relativePos = glm::vec3((pos[currBoid] - gridMin) * inverseCellWidth);
  // - Identify which cells may contain neighbors. This isn't always 8.

    // vars for the rules
    int numNeighbors1 = 0;      // different neighbor sum for rules 1 and 3 in case their neighbor dist is different
    int numNeighbors3 = 0;
    glm::vec3 perceived_center(0.f), c(0.f), perceived_vel(0.f);

    float cellOffset = maxRuleDist * inverseCellWidth;

    int minX = imax((int)relativePos.x - cellOffset, 0);
    int minY = imax((int)relativePos.y - cellOffset, 0);
    int minZ = imax((int)relativePos.z - cellOffset, 0);
    int maxX = imin((int)relativePos.x + cellOffset, gridResolution - 1);
    int maxY = imin((int)relativePos.y + cellOffset, gridResolution - 1);
    int maxZ = imin((int)relativePos.z + cellOffset, gridResolution - 1);

    for (int z = minZ; z <= maxZ; ++z)
    {
        for (int y = minY; y <= maxY; ++y)
        {
            for (int x = minX; x <= maxX; ++x)
            {
                glm::ivec3 neighborPos = glm::ivec3(x, y, z);
                int neighborGrid = gridIndex3Dto1D(neighborPos.x, neighborPos.y, neighborPos.z, gridResolution);
                // - For each cell, read the start/end indices in the boid pointer array.
                // - Access each boid in the cell and compute velocity change from
                //   the boids rules, if this boid is within the neighborhood distance.
                int start = gridCellStartIndices[neighborGrid];
                int end = gridCellEndIndices[neighborGrid];
                if (start < 0 || end < 0) continue;
                for (int i = start; i <= end; ++i)
                {
                    int boid = particleArrayIndices[i];
                    if (boid != currBoid) // don't want to check this boid
                    {
                        float dist = glm::distance(pos[currBoid], pos[boid]);
                        if (dist < rule1Distance)
                        {
                            perceived_center += pos[boid];
                            ++numNeighbors1;
                        }
                        if (dist < rule2Distance)
                        {
                            c -= (pos[boid] - pos[currBoid]);
                        }
                        if (dist < rule3Distance)
                        {
                            perceived_vel += vel1[boid];
                            ++numNeighbors3;
                        }
                    }
                }
            }
        }
    }
    // rule adjustments and summing up rule results
    glm::vec3 newVel = vel1[currBoid];
    if (numNeighbors1 > 0)
    {
        perceived_center /= numNeighbors1;
        newVel += (perceived_center - pos[currBoid]) * rule1Scale;
    }
    newVel += c * rule2Scale;
    if (numNeighbors3 > 0)
    {
        perceived_vel /= numNeighbors3;
        newVel += perceived_vel * rule3Scale;
    }
  // - Clamp the speed change before putting the new speed in vel2
    newVel = newVel.length() > maxSpeed ? glm::normalize(newVel) * maxSpeed : newVel;
    vel2[currBoid] = newVel;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2, float maxRuleDist) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= N) return;
    glm::vec3 relativePos = glm::vec3((pos[idx] - gridMin) * inverseCellWidth);
  // - Identify which cells may contain neighbors. This isn't always 8.
    // vars for the rules
    int numNeighbors1 = 0;      // different neighbor sum for rules 1 and 3 in case their neighbor dist is different
    int numNeighbors3 = 0;
    glm::vec3 perceived_center(0.f), c(0.f), perceived_vel(0.f);

    float cellOffset = maxRuleDist * inverseCellWidth;

    int minX = imax((int)relativePos.x - cellOffset, 0);
    int minY = imax((int)relativePos.y - cellOffset, 0);
    int minZ = imax((int)relativePos.z - cellOffset, 0);
    int maxX = imin((int)relativePos.x + cellOffset, gridResolution - 1);
    int maxY = imin((int)relativePos.y + cellOffset, gridResolution - 1);
    int maxZ = imin((int)relativePos.z + cellOffset, gridResolution - 1);
    for (int z = minZ; z <= maxZ; ++z)
    {
        for (int y = minY; y <= maxY; ++y)
        {
            for (int x = minX; x <= maxX; ++x)
            {
                glm::ivec3 neighborPos = glm::ivec3(x, y, z);
                int neighborGrid = gridIndex3Dto1D(neighborPos.x, neighborPos.y, neighborPos.z, gridResolution);
                // - For each cell, read the start/end indices in the boid pointer array.
                //   DIFFERENCE: For best results, consider what order the cells should be
                //   checked in to maximize the memory benefits of reordering the boids data.
                int start = gridCellStartIndices[neighborGrid];
                int end = gridCellEndIndices[neighborGrid];
                if (start < 0 || end < 0) continue;

                // - Access each boid in the cell and compute velocity change from
                for (int boid = start; boid <= end; ++boid)
                {
                    if (boid != idx) // don't want to check this boid
                    {
                        float dist = glm::distance(pos[idx], pos[boid]);
                        if (dist < rule1Distance)
                        {
                            perceived_center += pos[boid];
                            ++numNeighbors1;
                        }
                        if (dist < rule2Distance)
                        {
                            c -= (pos[boid] - pos[idx]);
                        }
                        if (dist < rule3Distance)
                        {
                            perceived_vel += vel1[boid];
                            ++numNeighbors3;
                        }
                    }
                }
            }
        }
    }
  //   the boids rules, if this boid is within the neighborhood distance.
    glm::vec3 newVel = vel1[idx];
    if (numNeighbors1 > 0)
    {
        perceived_center /= numNeighbors1;
        newVel += (perceived_center - pos[idx]) * rule1Scale;
    }
    newVel += c * rule2Scale;
    if (numNeighbors3 > 0)
    {
        perceived_vel /= numNeighbors3;
        newVel += perceived_vel * rule3Scale;
    }
  // - Clamp the speed change before putting the new speed in vel2
    newVel = newVel.length() > maxSpeed ? glm::normalize(newVel) * maxSpeed : newVel;
    vel2[idx] = newVel;
}

// Helper kernel to sort our positions and velocities. This method worked over thrust
__global__ void kernelSortArrays(int N, glm::vec3* pos, glm::vec3* sorted_pos, glm::vec3* vel, glm::vec3* sorted_vel, int* particleArrayIndices) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < N) {
        sorted_pos[idx] = pos[particleArrayIndices[idx]];
        sorted_vel[idx] = vel[particleArrayIndices[idx]];
    }
}
// Helper kernel to unsort our positions, making it work for the next loop
__global__ void kernelUnsortArrays(int N, glm::vec3* pos, glm::vec3* sorted_pos, glm::vec3* vel, glm::vec3* sorted_vel, int* particleArrayIndices) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < N) {
        pos[particleArrayIndices[idx]] = sorted_pos[idx];
        vel[particleArrayIndices[idx]] = sorted_vel[idx];
    }
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
    kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
    kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
  // TODO-1.2 ping-pong the velocity buffers
    hipMemcpy(dev_vel1, dev_vel2, sizeof(float) * 3 * numObjects, hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
    // Wrap device vectors in thrust iterators for use with thrust.
    dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
    dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    // first we tag all empty cells as -1
    kernResetIntBuffer << <((gridCellCount + blockSize - 1) / blockSize), blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    //kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1); // no need to also do end, sinze start -1 already signals
    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
                                                                                dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2, maxRuleDist);
  // - Update positions
    kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
  // - Ping-pong buffers as needed
    hipMemcpy(dev_vel1, dev_vel2, sizeof(float) * 3 * numObjects, hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grid
    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
    dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
    dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    // first we tag all empty cells as -1
    kernResetIntBuffer << <((gridCellCount + blockSize - 1) / blockSize), blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    //kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);
    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
    kernelSortArrays << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, dev_sortedPos, dev_vel1, dev_sortedVel1, dev_particleArrayIndices);
  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
                                                                                dev_gridCellStartIndices, dev_gridCellEndIndices, dev_sortedPos, dev_sortedVel1, dev_sortedVel2, maxRuleDist);
  // - Update positions
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_sortedPos, dev_sortedVel2);
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
    kernelUnsortArrays << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, dev_sortedPos, dev_vel2, dev_sortedVel2, dev_particleArrayIndices);
    hipMemcpy(dev_vel1, dev_vel2, sizeof(float) * 3 * numObjects, hipMemcpyDeviceToDevice);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_sortedPos);
  hipFree(dev_sortedVel1);
  hipFree(dev_sortedVel2);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}