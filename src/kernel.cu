#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA

#include <hip/hip_runtime.h>
#include "kernel.h"
#include "utilityCore.hpp"

#include <cmath>
#include <cstdio>
#include <iostream>
#include <vector>

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>

#include <glm/glm.hpp>

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 1024

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 2.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

glm::vec3* dev_pos_sorted;
glm::vec3* dev_vel1_sorted;
glm::vec3* dev_vel2_sorted;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  // Grid-Looping Optimization: Here change to 1��distance, Dynamic boundary calculation works better with smaller cells.
  // When cell_width = 2��distance, saves only 0-2 cells (8��6-8); when cell_width = 1��distance, saves 12-19 cells (27��8-15).
  // Optimization overhead is justified only when grid cells are small enough to create significant search space reduction.
  gridCellWidth = 1.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  // Setup thrust pointers for sorting
  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
  hipMalloc((void**)&dev_pos_sorted, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos_sorted failed!");

  hipMalloc((void**)&dev_vel1_sorted, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1_sorted failed!");

  hipMalloc((void**)&dev_vel2_sorted, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2_sorted failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3* pos, const glm::vec3* vel) {
    glm::vec3 rule1_velocity = glm::vec3(0.0f);
    glm::vec3 rule2_velocity = glm::vec3(0.0f);
    glm::vec3 rule3_velocity = glm::vec3(0.0f);

    glm::vec3 myPos = pos[iSelf];

    int rule1_neighbors = 0;
    int rule3_neighbors = 0;

    // Loop through all other boids
    for (int i = 0; i < N; i++) {
        if (i == iSelf) continue; // Skip self

        glm::vec3 otherPos = pos[i];
        float distance = glm::length(myPos - otherPos);

        // Rule 1: Cohesion - boids fly towards their local perceived center of mass
        if (distance < rule1Distance) {
            rule1_velocity += otherPos;
            rule1_neighbors++;
        }

        // Rule 2: Separation - boids try to stay a distance d away from each other
        if (distance < rule2Distance) {
            rule2_velocity -= (otherPos - myPos);
        }

        // Rule 3: Alignment - boids try to match velocity with near boids
        if (distance < rule3Distance) {
            rule3_velocity += vel[i];
            rule3_neighbors++;
        }
    }

    // Apply the rules
    if (rule1_neighbors > 0) {
        rule1_velocity /= rule1_neighbors; // Get average position
        rule1_velocity = (rule1_velocity - myPos) * rule1Scale; // Direction towards center
    }

    rule2_velocity *= rule2Scale;

    if (rule3_neighbors > 0) {
        rule3_velocity /= rule3_neighbors; // Get average velocity
        rule3_velocity *= rule3Scale;
    }

    return rule1_velocity + rule2_velocity + rule3_velocity;
}


/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3* pos,
    glm::vec3* vel1, glm::vec3* vel2) {

    // Get the index of the current thread
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    // Compute velocity change based on the three rules
    glm::vec3 velocityChange = computeVelocityChange(N, index, pos, vel1);

    // Update velocity
    glm::vec3 newVelocity = vel1[index] + velocityChange;

    // Clamp the speed to maxSpeed
    float speed = glm::length(newVelocity);
    if (speed > maxSpeed) {
        newVelocity = (newVelocity / speed) * maxSpeed;
    }

    // Record the new velocity into vel2
    vel2[index] = newVelocity;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
    glm::vec3 gridMin, float inverseCellWidth,
    glm::vec3* pos, int* indices, int* gridIndices) {

    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    // Set up parallel array of integer indices as pointers to the actual boid data
    indices[index] = index;

    // Label each boid with the index of its grid cell
    glm::vec3 boidPos = pos[index];

    // Convert world position to grid coordinates
    glm::vec3 gridPos = (boidPos - gridMin) * inverseCellWidth;

    // Convert to integer grid coordinates (with bounds checking)
    int gridX = (int)gridPos.x;
    int gridY = (int)gridPos.y;
    int gridZ = (int)gridPos.z;

    // Clamp to valid grid range
    gridX = gridX < 0 ? 0 : (gridX >= gridResolution ? gridResolution - 1 : gridX);
    gridY = gridY < 0 ? 0 : (gridY >= gridResolution ? gridResolution - 1 : gridY);
    gridZ = gridZ < 0 ? 0 : (gridZ >= gridResolution ? gridResolution - 1 : gridZ);

    // Convert 3D grid coordinates to 1D index
    int gridIndex = gridIndex3Dto1D(gridX, gridY, gridZ, gridResolution);

    gridIndices[index] = gridIndex;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    int currentGridIndex = particleGridIndices[index];

    // Check if this is the start of a new cell
    if (index == 0 || particleGridIndices[index - 1] != currentGridIndex) {
        gridCellStartIndices[currentGridIndex] = index;
    }

    // Check if this is the end of a cell
    if (index == N - 1 || particleGridIndices[index + 1] != currentGridIndex) {
        gridCellEndIndices[currentGridIndex] = index;
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2


    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    // Get current boid's data
    glm::vec3 myPos = pos[index];

    // Initialize rule velocities
    glm::vec3 rule1_velocity = glm::vec3(0.0f);
    glm::vec3 rule2_velocity = glm::vec3(0.0f);
    glm::vec3 rule3_velocity = glm::vec3(0.0f);

    int rule1_neighbors = 0;
    int rule3_neighbors = 0;

    // Extra Credit : Grid-Looping Optimization: Calculate dynamic search bounds
    float maxNeighborDistance = fmaxf(fmaxf(rule1Distance, rule2Distance), rule3Distance);

    // Calculate bounding box for neighbor search
    glm::vec3 minBound = myPos - glm::vec3(maxNeighborDistance);
    glm::vec3 maxBound = myPos + glm::vec3(maxNeighborDistance);

    // Convert to grid coordinates
    glm::vec3 minGridCoord = (minBound - gridMin) * inverseCellWidth;
    glm::vec3 maxGridCoord = (maxBound - gridMin) * inverseCellWidth;

    // Convert to integer grid indices with bounds checking
    int minGridX = max(0, (int)minGridCoord.x);
    int maxGridX = min(gridResolution - 1, (int)maxGridCoord.x);
    int minGridY = max(0, (int)minGridCoord.y);
    int maxGridY = min(gridResolution - 1, (int)maxGridCoord.y);
    int minGridZ = max(0, (int)minGridCoord.z);
    int maxGridZ = min(gridResolution - 1, (int)maxGridCoord.z);

    // Dynamic loop - only check cells that could contain neighbors
    for (int gz = minGridZ; gz <= maxGridZ; gz++) {
        for (int gy = minGridY; gy <= maxGridY; gy++) {
            for (int gx = minGridX; gx <= maxGridX; gx++) {

                int neighborGridIndex = gridIndex3Dto1D(gx, gy, gz, gridResolution);

                int startIndex = gridCellStartIndices[neighborGridIndex];
                int endIndex = gridCellEndIndices[neighborGridIndex];

                // Skip empty cells
                if (startIndex == -1 || endIndex == -1) {
                    continue;
                }

                // Check all boids in this neighboring cell
                for (int i = startIndex; i <= endIndex; i++) {
                    int neighborBoidIndex = particleArrayIndices[i];

                    if (neighborBoidIndex == index) continue; // Skip self

                    glm::vec3 otherPos = pos[neighborBoidIndex];
                    float distance = glm::length(myPos - otherPos);

                    // Apply the three rules with distance checks
                    if (distance < rule1Distance) {
                        rule1_velocity += otherPos;
                        rule1_neighbors++;
                    }

                    if (distance < rule2Distance) {
                        rule2_velocity -= (otherPos - myPos);
                    }

                    if (distance < rule3Distance) {
                        rule3_velocity += vel1[neighborBoidIndex];
                        rule3_neighbors++;
                    }
                }
            }
        }
    }

    // Apply the rules (same logic as before)
    if (rule1_neighbors > 0) {
        rule1_velocity /= rule1_neighbors;
        rule1_velocity = (rule1_velocity - myPos) * rule1Scale;
    }

    rule2_velocity *= rule2Scale;

    if (rule3_neighbors > 0) {
        rule3_velocity /= rule3_neighbors;
        rule3_velocity *= rule3Scale;
    }

    glm::vec3 velocityChange = rule1_velocity + rule2_velocity + rule3_velocity;
    glm::vec3 newVelocity = vel1[index] + velocityChange;

    // Clamp speed
    float speed = glm::length(newVelocity);
    if (speed > maxSpeed) {
        newVelocity = (newVelocity / speed) * maxSpeed;
    }

    vel2[index] = newVelocity;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    // Get current boid's data (now directly from sorted arrays)
    glm::vec3 myPos = pos[index];

    // Initialize rule velocities
    glm::vec3 rule1_velocity = glm::vec3(0.0f);
    glm::vec3 rule2_velocity = glm::vec3(0.0f);
    glm::vec3 rule3_velocity = glm::vec3(0.0f);

    int rule1_neighbors = 0;
    int rule3_neighbors = 0;

    // Extra Credit : Grid-Looping Optimization Calculate dynamic search bounds
    float maxNeighborDistance = fmaxf(fmaxf(rule1Distance, rule2Distance), rule3Distance);

    // Calculate bounding box for neighbor search
    glm::vec3 minBound = myPos - glm::vec3(maxNeighborDistance);
    glm::vec3 maxBound = myPos + glm::vec3(maxNeighborDistance);

    // Convert to grid coordinates
    glm::vec3 minGridCoord = (minBound - gridMin) * inverseCellWidth;
    glm::vec3 maxGridCoord = (maxBound - gridMin) * inverseCellWidth;

    // Convert to integer grid indices with bounds checking
    int minGridX = max(0, (int)minGridCoord.x);
    int maxGridX = min(gridResolution - 1, (int)maxGridCoord.x);
    int minGridY = max(0, (int)minGridCoord.y);
    int maxGridY = min(gridResolution - 1, (int)maxGridCoord.y);
    int minGridZ = max(0, (int)minGridCoord.z);
    int maxGridZ = min(gridResolution - 1, (int)maxGridCoord.z);

    // Dynamic loop with memory-efficient order (z-y-x for better cache locality)
    for (int gz = minGridZ; gz <= maxGridZ; gz++) {
        for (int gy = minGridY; gy <= maxGridY; gy++) {
            for (int gx = minGridX; gx <= maxGridX; gx++) {

                int neighborGridIndex = gridIndex3Dto1D(gx, gy, gz, gridResolution);

                int startIndex = gridCellStartIndices[neighborGridIndex];
                int endIndex = gridCellEndIndices[neighborGridIndex];

                // Skip empty cells
                if (startIndex == -1 || endIndex == -1) {
                    continue;
                }

                // Check all boids in this neighboring cell
                for (int i = startIndex; i <= endIndex; i++) {
                    if (i == index) continue; // Skip self

                    glm::vec3 otherPos = pos[i];
                    float distance = glm::length(myPos - otherPos);

                    // Apply the three rules with distance checks
                    if (distance < rule1Distance) {
                        rule1_velocity += otherPos;
                        rule1_neighbors++;
                    }

                    if (distance < rule2Distance) {
                        rule2_velocity -= (otherPos - myPos);
                    }

                    if (distance < rule3Distance) {
                        rule3_velocity += vel1[i];
                        rule3_neighbors++;
                    }
                }
            }
        }
    }

    // Apply the rules
    if (rule1_neighbors > 0) {
        rule1_velocity /= rule1_neighbors;
        rule1_velocity = (rule1_velocity - myPos) * rule1Scale;
    }

    rule2_velocity *= rule2Scale;

    if (rule3_neighbors > 0) {
        rule3_velocity /= rule3_neighbors;
        rule3_velocity *= rule3Scale;
    }

    glm::vec3 velocityChange = rule1_velocity + rule2_velocity + rule3_velocity;
    glm::vec3 newVelocity = vel1[index] + velocityChange;

    // Clamp speed
    float speed = glm::length(newVelocity);
    if (speed > maxSpeed) {
        newVelocity = (newVelocity / speed) * maxSpeed;
    }

    vel2[index] = newVelocity;
}




__global__ void kernReshuffleData(int N, int* particleArrayIndices,
    glm::vec3* pos_unsorted, glm::vec3* vel_unsorted,
    glm::vec3* pos_sorted, glm::vec3* vel_sorted) {

    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    // Get the original boid index from the sorted array
    int originalIndex = particleArrayIndices[index];

    // Copy data from unsorted to sorted arrays
    pos_sorted[index] = pos_unsorted[originalIndex];
    vel_sorted[index] = vel_unsorted[originalIndex];
}



/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

    // Update velocities using brute force neighbor search
    kernUpdateVelocityBruteForce << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");

    // Update positions based on new velocities
    kernUpdatePos << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");

    // Ping-pong the velocity buffers
    glm::vec3* temp = dev_vel1;
    dev_vel1 = dev_vel2;
    dev_vel2 = temp;
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed


    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    dim3 fullBlocksPerGridCell((gridCellCount + blockSize - 1) / blockSize);

    // Label each particle with its array index and grid index
    kernComputeIndices << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
        dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    checkCUDAErrorWithLine("kernComputeIndices failed!");

    //  Reset grid cell start/end indices
    kernResetIntBuffer << <fullBlocksPerGridCell, threadsPerBlock >> > (
        gridCellCount, dev_gridCellStartIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer start failed!");

    kernResetIntBuffer << <fullBlocksPerGridCell, threadsPerBlock >> > (
        gridCellCount, dev_gridCellEndIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer end failed!");

    //  Sort particles by grid index using Thrust
    thrust::sort_by_key(dev_thrust_particleGridIndices,
        dev_thrust_particleGridIndices + numObjects,
        dev_thrust_particleArrayIndices);

    //  Identify start and end indices for each cell
    kernIdentifyCellStartEnd << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, dev_particleGridIndices,
        dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

    //  Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, gridCellWidth,
        dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices,
        dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");

    //  Update positions
    kernUpdatePos << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");

    //  Ping-pong buffers
    glm::vec3* temp = dev_vel1;
    dev_vel1 = dev_vel2;
    dev_vel2 = temp;
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.


    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    dim3 fullBlocksPerGridCell((gridCellCount + blockSize - 1) / blockSize);

    //  Label each particle with its array index and grid index
    kernComputeIndices << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
        dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    checkCUDAErrorWithLine("kernComputeIndices failed!");

    //  Reset grid cell start/end indices
    kernResetIntBuffer << <fullBlocksPerGridCell, threadsPerBlock >> > (
        gridCellCount, dev_gridCellStartIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer start failed!");

    kernResetIntBuffer << <fullBlocksPerGridCell, threadsPerBlock >> > (
        gridCellCount, dev_gridCellEndIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer end failed!");

    // Sort particles by grid index using Thrust
    thrust::sort_by_key(dev_thrust_particleGridIndices,
        dev_thrust_particleGridIndices + numObjects,
        dev_thrust_particleArrayIndices);

    // Identify start and end indices for each cell
    kernIdentifyCellStartEnd << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, dev_particleGridIndices,
        dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

    // BIG DIFFERENCE - Reshuffle the particle data for coherent access
    kernReshuffleData << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, dev_particleArrayIndices,
        dev_pos, dev_vel1,
        dev_pos_sorted, dev_vel1_sorted);
    checkCUDAErrorWithLine("kernReshuffleData failed!");

    // Perform velocity updates using coherent neighbor search
    kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, gridCellWidth,
        dev_gridCellStartIndices, dev_gridCellEndIndices,
        dev_pos_sorted, dev_vel1_sorted, dev_vel2_sorted);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchCoherent failed!");

    //  Update positions using sorted data
    kernUpdatePos << <fullBlocksPerGrid, threadsPerBlock >> > (
        numObjects, dt, dev_pos_sorted, dev_vel2_sorted);
    checkCUDAErrorWithLine("kernUpdatePos failed!");

    // Ping-pong buffers

    glm::vec3* temp_pos = dev_pos;
    dev_pos = dev_pos_sorted;
    dev_pos_sorted = temp_pos;

    glm::vec3* temp_vel = dev_vel1;
    dev_vel1 = dev_vel2_sorted;
    dev_vel2_sorted = dev_vel1_sorted;
    dev_vel1_sorted = temp_vel;
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);


  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_pos_sorted);
  hipFree(dev_vel1_sorted);
  hipFree(dev_vel2_sorted);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
