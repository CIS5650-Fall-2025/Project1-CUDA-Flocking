#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/sort.h>

#include <cmath>
#include <cstdio>
#include <glm/glm.hpp>
#include <iostream>
#include <vector>

#include "kernel.h"
#include "utilityCore.hpp"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax(a, b) (((a) > (b)) ? (a) : (b))
#endif

#ifndef imin
#define imin(a, b) (((a) < (b)) ? (a) : (b))
#endif

#define clampGrid(x, gridRes) imin(imax(x, 0), gridRes - 1)

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
 * Check for CUDA errors; print and exit if there was a problem.
 */
void checkCUDAError(const char* msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

/*****************
 * Configuration *
 *****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 7.5f
#define rule2Distance 3.0f
#define rule3Distance 5.5f

#define rule1Scale 0.05f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
 * Kernel state (pointers are device pointers) *
 ***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3* dev_pos;
glm::vec3* dev_vel1;
glm::vec3* dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int* dev_particleArrayIndices;  // What index in dev_pos and dev_velX represents
// this particle?
int* dev_particleGridIndices;  // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int* dev_gridCellStartIndices;  // What part of dev_particleArrayIndices belongs
int* dev_gridCellEndIndices;    // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
thrust::device_ptr<glm::vec3> dev_thrust_pos;
thrust::device_ptr<glm::vec3> dev_thrust_vel1;
thrust::device_ptr<glm::vec3> dev_thrust_vel2;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
 * initSimulation *
 ******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
 * LOOK-1.2 - this is a typical helper function for a CUDA kernel.
 * Function for generating a random vec3.
 */
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng),
                   (float)unitDistrib(rng));
}

/**
 * LOOK-1.2 - This is a basic CUDA kernel.
 * CUDA kernel for generating boids with a specified mass randomly around the
 * star.
 */
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3* arr,
                                           float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
 * Initialize memory, update some globals
 */
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(
      1, numObjects, dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth =
      2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  dev_thrust_pos = thrust::device_pointer_cast(dev_pos);
  dev_thrust_vel1 = thrust::device_pointer_cast(dev_vel1);
  dev_thrust_vel2 = thrust::device_pointer_cast(dev_vel2);

  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  dev_thrust_particleArrayIndices =
      thrust::device_pointer_cast(dev_particleArrayIndices);

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  dev_thrust_particleGridIndices =
      thrust::device_pointer_cast(dev_particleGridIndices);

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipDeviceSynchronize();
}

/******************
 * copyBoidsToVBO *
 ******************/

/**
 * Copy the boid positions into the VBO so that they can be drawn by OpenGL.
 */
__global__ void kernCopyPositionsToVBO(int N, glm::vec3* pos, float* vbo,
                                       float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3* vel, float* vbo,
                                        float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
 * Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
 */
void Boids::copyBoidsToVBO(float* vbodptr_positions,
                           float* vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_vel1, vbodptr_velocities, scene_scale);
  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}

/******************
 * stepSimulation *
 ******************/

/**
 * LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
 * __device__ code can be called from a __global__ context
 * Compute the new velocity on the body with index `iSelf` due to the `N` boids
 * in the `pos` and `vel` arrays.
 */
__device__ glm::vec3 computeVelocityChange(int N, int iSelf,
                                           const glm::vec3* pos,
                                           const glm::vec3* vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which
  // excludes themselves Rule 2: boids try to stay a distance d away from each
  // other Rule 3: boids try to match the speed of surrounding boids

  // Velocity change to return
  glm::vec3 velChange(0, 0, 0);

  // Rule 1 initialization
  unsigned int numNeighborsRule1 = 0;
  glm::vec3 perceivedCenter(0, 0, 0);

  // Rule 2 initialization
  glm::vec3 c(0, 0, 0);

  // Rule 3 initialization
  unsigned int numNeighborsRule3 = 0;
  glm::vec3 perceivedVel(0, 0, 0);

  for (unsigned int iOther = 0; iOther < N; iOther++) {
    if (iOther != iSelf) {
      float distance = glm::distance(pos[iOther], pos[iSelf]);
      if (distance < rule1Distance) {
        perceivedCenter += pos[iOther];
        numNeighborsRule1++;
      }
      if (distance < rule2Distance) {
        c -= (pos[iOther] - pos[iSelf]);
      }
      if (distance < rule3Distance) {
        perceivedVel += vel[iOther];
        numNeighborsRule3++;
      }
    }
  }

  // Rule 1 velocity change
  numNeighborsRule1 += (numNeighborsRule1 == 0);
  perceivedCenter /= numNeighborsRule1;
  velChange += (perceivedCenter - pos[iSelf]) * rule1Scale;

  // Rule 2 velocity change
  velChange += (c * rule2Scale);

  // Rule 3 velocity change
  numNeighborsRule3 += (numNeighborsRule3 == 0);
  perceivedVel /= numNeighborsRule3;
  velChange += (perceivedVel - vel[iSelf]) * rule3Scale;

  return velChange;
}

/**
 * TODO-1.2 implement basic flocking
 * For each of the `N` bodies, update its position based on its current
 * velocity.
 */
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3* pos,
                                             glm::vec3* vel1, glm::vec3* vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < N) {
    glm::vec3 newVel = vel1[index] + computeVelocityChange(N, index, pos, vel1);
    vel2[index] = glm::clamp(newVel, -maxSpeed, maxSpeed);
  }
}

/**
 * LOOK-1.2 Since this is pretty trivial, we implemented it for you.
 * For each of the `N` bodies, update its position based on its current
 * velocity.
 */
__global__ void kernUpdatePos(int N, float dt, glm::vec3* pos, glm::vec3* vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution, glm::vec3 gridMin,
                                   float inverseCellWidth, glm::vec3* pos,
                                   int* indices, int* gridIndices) {
  // TODO-2.1
  // - Label each boid with the index of its grid cell.
  // - Set up a parallel array of integer indices as pointers to the actual
  //   boid data in pos and vel1/vel2
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < N) {
    indices[index] = index;
    glm::vec3 iPos = pos[index];
    int iX = (int)floorf((iPos.x - gridMin.x) * inverseCellWidth);
    int iY = (int)floorf((iPos.y - gridMin.y) * inverseCellWidth);
    int iZ = (int)floorf((iPos.z - gridMin.z) * inverseCellWidth);
    iX = clampGrid(iX, gridResolution);
    iY = clampGrid(iY, gridResolution);
    iZ = clampGrid(iZ, gridResolution);
    gridIndices[index] = gridIndex3Dto1D(iX, iY, iZ, gridResolution);
  }
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int* intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int* particleGridIndices,
                                         int* gridCellStartIndices,
                                         int* gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < N) {
    int gridIndex = particleGridIndices[index];
    if (index == 0) {  // First element must be a start
      gridCellStartIndices[gridIndex] = 0;
    } else {
      // If the previous boid is in a different grid, gridIndex is both
      // the start of gridIndex and the end of prevGridIndex (non-inclusive)
      int prevGridIndex = particleGridIndices[index - 1];
      if (gridIndex != prevGridIndex) {
        gridCellStartIndices[gridIndex] = index;
        gridCellEndIndices[prevGridIndex] = index;
      }
    }
    if (index == N - 1) {  // Last element must be an end
      gridCellEndIndices[gridIndex] = N;
    }
  }
}

__global__ void kernUpdateVelNeighborSearchScattered(
    int N, int gridResolution, glm::vec3 gridMin, float inverseCellWidth,
    float cellWidth, int* gridCellStartIndices, int* gridCellEndIndices,
    int* particleArrayIndices, glm::vec3* pos, glm::vec3* vel1,
    glm::vec3* vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < N) {
    // The following line is not needed, since index is unique
    // int iSelf = particleArrayIndices[index];
    int iSelf = index;
    glm::vec3 iPos = pos[iSelf];

    int iX = (int)floorf((iPos.x - gridMin.x) * inverseCellWidth);
    int iY = (int)floorf((iPos.y - gridMin.y) * inverseCellWidth);
    int iZ = (int)floorf((iPos.z - gridMin.z) * inverseCellWidth);

    iX = clampGrid(iX, gridResolution);
    iY = clampGrid(iY, gridResolution);
    iZ = clampGrid(iZ, gridResolution);

    // Compute velocity change only with boids in neighboring cells
    glm::vec3 velChange(0, 0, 0);

    // Rule 1 initialization
    unsigned int numNeighborsRule1 = 0;
    glm::vec3 perceivedCenter(0, 0, 0);

    // Rule 2 initialization
    glm::vec3 c(0, 0, 0);

    // Rule 3 initialization
    unsigned int numNeighborsRule3 = 0;
    glm::vec3 perceivedVel(0, 0, 0);

    // Compute an array of neighboring grid cell indices
    // Looping in the order z, y, x is the most efficient, since memory is
    // contiguous along x, then y, then z
    for (unsigned int z = clampGrid(iZ - 1, gridResolution);
         z <= clampGrid(iZ + 1, gridResolution); z++) {
      for (unsigned int y = clampGrid(iY - 1, gridResolution);
           y <= clampGrid(iY + 1, gridResolution); y++) {
        for (unsigned int x = clampGrid(iX - 1, gridResolution);
             x <= clampGrid(iX + 1, gridResolution); x++) {
          int neighborIndex = gridIndex3Dto1D(x, y, z, gridResolution);
          int startIndex = gridCellStartIndices[neighborIndex];
          int endIndex = gridCellEndIndices[neighborIndex];
          if (startIndex != -1 && endIndex != -1) {
            for (unsigned int j = startIndex; j < endIndex; j++) {
              int iOther = particleArrayIndices[j];
              if (iOther != iSelf) {
                float distance = glm::distance(pos[iOther], pos[iSelf]);
                if (distance < rule1Distance) {
                  perceivedCenter += pos[iOther];
                  numNeighborsRule1++;
                }
                if (distance < rule2Distance) {
                  c -= (pos[iOther] - pos[iSelf]);
                }
                if (distance < rule3Distance) {
                  perceivedVel += vel1[iOther];
                  numNeighborsRule3++;
                }
              }
            }
          }
        }
      }
    }

    // Rule 1 velocity change
    numNeighborsRule1 += (numNeighborsRule1 == 0);
    perceivedCenter /= numNeighborsRule1;
    velChange += (perceivedCenter - pos[iSelf]) * rule1Scale;

    // Rule 2 velocity change
    velChange += (c * rule2Scale);

    // Rule 3 velocity change
    numNeighborsRule3 += (numNeighborsRule3 == 0);
    perceivedVel /= numNeighborsRule3;
    velChange += (perceivedVel - vel1[iSelf]) * rule3Scale;

    // Update velocity
    glm::vec3 newVel = vel1[iSelf] + velChange;
    vel2[iSelf] = glm::clamp(newVel, -maxSpeed, maxSpeed);
  }
}

__global__ void kernUpdateVelNeighborSearchCoherent(
    int N, int gridResolution, glm::vec3 gridMin, float inverseCellWidth,
    float cellWidth, int* gridCellStartIndices, int* gridCellEndIndices,
    glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
  // TODO-2.3 - This should be very similar to
  // kernUpdateVelNeighborSearchScattered, except with one less level of
  // indirection. This should expect gridCellStartIndices and gridCellEndIndices
  // to refer directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < N) {
    int iSelf = index;
    glm::vec3 iPos = pos[iSelf];

    int iX = (int)floorf((iPos.x - gridMin.x) * inverseCellWidth);
    int iY = (int)floorf((iPos.y - gridMin.y) * inverseCellWidth);
    int iZ = (int)floorf((iPos.z - gridMin.z) * inverseCellWidth);

    iX = clampGrid(iX, gridResolution);
    iY = clampGrid(iY, gridResolution);
    iZ = clampGrid(iZ, gridResolution);

    // Compute velocity change only with boids in neighboring cells
    glm::vec3 velChange(0, 0, 0);

    // Rule 1 initialization
    unsigned int numNeighborsRule1 = 0;
    glm::vec3 perceivedCenter(0, 0, 0);

    // Rule 2 initialization
    glm::vec3 c(0, 0, 0);

    // Rule 3 initialization
    unsigned int numNeighborsRule3 = 0;
    glm::vec3 perceivedVel(0, 0, 0);

    // Compute an array of neighboring grid cell indices
    // Looping in the order z, y, x is the most efficient, since memory is
    // contiguous along x, then y, then z
    for (unsigned int z = clampGrid(iZ - 1, gridResolution);
         z <= clampGrid(iZ + 1, gridResolution); z++) {
      for (unsigned int y = clampGrid(iY - 1, gridResolution);
           y <= clampGrid(iY + 1, gridResolution); y++) {
        for (unsigned int x = clampGrid(iX - 1, gridResolution);
             x <= clampGrid(iX + 1, gridResolution); x++) {
          int neighborIndex = gridIndex3Dto1D(x, y, z, gridResolution);
          int startIndex = gridCellStartIndices[neighborIndex];
          int endIndex = gridCellEndIndices[neighborIndex];
          if (startIndex != -1 && endIndex != -1) {
            for (unsigned int iOther = startIndex; iOther < endIndex;
                 iOther++) {
              if (iOther != iSelf) {
                float distance = glm::distance(pos[iOther], pos[iSelf]);
                if (distance < rule1Distance) {
                  perceivedCenter += pos[iOther];
                  numNeighborsRule1++;
                }
                if (distance < rule2Distance) {
                  c -= (pos[iOther] - pos[iSelf]);
                }
                if (distance < rule3Distance) {
                  perceivedVel += vel1[iOther];
                  numNeighborsRule3++;
                }
              }
            }
          }
        }
      }
    }

    // Rule 1 velocity change
    numNeighborsRule1 += (numNeighborsRule1 == 0);
    perceivedCenter /= numNeighborsRule1;
    velChange += (perceivedCenter - pos[iSelf]) * rule1Scale;

    // Rule 2 velocity change
    velChange += (c * rule2Scale);

    // Rule 3 velocity change
    numNeighborsRule3 += (numNeighborsRule3 == 0);
    perceivedVel /= numNeighborsRule3;
    velChange += (perceivedVel - vel1[iSelf]) * rule3Scale;

    // Update velocity
    glm::vec3 newVel = vel1[iSelf] + velChange;
    vel2[iSelf] = glm::clamp(newVel, -maxSpeed, maxSpeed);
  }
}

/**
 * Step the entire N-body simulation by `dt` seconds.
 */
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in
  // time.
  // TODO-1.2 ping-pong the velocity buffers
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine(
      "stepSimulationNaive kernUpdateVelocityBruteForce failed!");
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos,
                                                  dev_vel2);
  checkCUDAErrorWithLine("stepSimulationNaive kernUpdatePos failed!");
  hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3),
             hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos,
      dev_particleArrayIndices, dev_particleGridIndices);
  checkCUDAErrorWithLine(
      "stepSimulationScatteredGrid kernComputeIndices failed!");

  thrust::sort_by_key(thrust::device, dev_thrust_particleGridIndices,
                      dev_thrust_particleGridIndices + numObjects,
                      dev_thrust_particleArrayIndices);

  kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(
      gridCellCount, dev_gridCellStartIndices, -1);
  checkCUDAErrorWithLine(
      "stepSimulationScatteredGrid kernResetIntBuffer dev_gridCellStartIndices "
      "failed!");

  kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(
      gridCellCount, dev_gridCellEndIndices, -1);
  checkCUDAErrorWithLine(
      "stepSimulationScatteredGrid kernResetIntBuffer dev_gridCellEndIndices "
      "failed!");

  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_particleGridIndices, dev_gridCellStartIndices,
      dev_gridCellEndIndices);
  checkCUDAErrorWithLine(
      "stepSimulationScatteredGrid kernIdentifyCellStartEnd failed!");

  kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
      gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
      dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine(
      "stepSimulationScatteredGrid kernUpdateVelNeighborSearchScattered "
      "failed!");

  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos,
                                                  dev_vel2);
  checkCUDAErrorWithLine("stepSimulationScatteredGrid kernUpdatePos failed!");

  hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3),
             hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos,
      dev_particleArrayIndices, dev_particleGridIndices);
  checkCUDAErrorWithLine(
      "stepSimulationCoherentGrid kernComputeIndices failed!");

  // Sort based on grid indices
  thrust::sort_by_key(
      thrust::device, dev_thrust_particleGridIndices,
      dev_thrust_particleGridIndices + numObjects,
      thrust::make_zip_iterator(dev_thrust_particleArrayIndices, dev_thrust_pos,
                                dev_thrust_vel1));

  kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(
      gridCellCount, dev_gridCellStartIndices, -1);
  checkCUDAErrorWithLine(
      "stepSimulationCoherentGrid kernResetIntBuffer dev_gridCellStartIndices "
      "failed!");

  kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(
      gridCellCount, dev_gridCellEndIndices, -1);
  checkCUDAErrorWithLine(
      "stepSimulationCoherentGrid kernResetIntBuffer dev_gridCellEndIndices "
      "failed!");

  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_particleGridIndices, dev_gridCellStartIndices,
      dev_gridCellEndIndices);
  checkCUDAErrorWithLine(
      "stepSimulationCoherentGrid kernIdentifyCellStartEnd failed!");

  kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
      gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_pos,
      dev_vel1, dev_vel2);
  checkCUDAErrorWithLine(
      "stepSimulationCoherentGrid kernUpdateVelNeighborSearchCoherent failed!");

  // Unsort to restore original order
  thrust::sort_by_key(
      thrust::device, dev_thrust_particleArrayIndices,
      dev_thrust_particleArrayIndices + numObjects,
      thrust::make_zip_iterator(dev_thrust_pos, dev_thrust_vel2));

  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos,
                                                  dev_vel2);
  checkCUDAErrorWithLine("stepSimulationCoherentGrid kernUpdatePos failed!");

  hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3),
             hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int* dev_intKeys;
  int* dev_intValues;
  int N = 10;

  std::unique_ptr<int[]> intKeys{new int[N]};
  std::unique_ptr<int[]> intValues{new int[N]};

  intKeys[0] = 0;
  intValues[0] = 0;
  intKeys[1] = 1;
  intValues[1] = 1;
  intKeys[2] = 0;
  intValues[2] = 2;
  intKeys[3] = 3;
  intValues[3] = 3;
  intKeys[4] = 0;
  intValues[4] = 4;
  intKeys[5] = 2;
  intValues[5] = 5;
  intKeys[6] = 2;
  intValues[6] = 6;
  intKeys[7] = 0;
  intValues[7] = 7;
  intKeys[8] = 5;
  intValues[8] = 8;
  intKeys[9] = 6;
  intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N,
             hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N,
             hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N,
             hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N,
             hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
